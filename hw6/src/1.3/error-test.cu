#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include "error_checks.h" // Macros CUDA_CHECK and CHECK_ERROR_MSG


__global__ void vector_add(int* x,double *C, const double *A, const double *B, int N)
{
    // Add the kernel code
    int idx = *x * blockIdx.x * blockDim.x + threadIdx.x;

    // Do not try to access past the allocated memory
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}


int main(void)
{
    const int N = 20;
    const int ThreadsInBlock = 128;
    double *dA, *dB, *dC;
    double hA[N], hB[N], hC[N];
    int *x = (int*)malloc(sizeof(int));
    *x = 1;

    for(int i = 0; i < N; ++i) {
        hA[i] = (double) i;
        hB[i] = (double) i * i;
    }

    /* 
       Add memory allocations and copies. Wrap your runtime function
       calls with CUDA_CHECK( ) macro
    */
    CUDA_CHECK(hipMalloc((void**)&dA, sizeof(double)*N));
    CUDA_CHECK(hipMalloc((void**)&dB, sizeof(double)*N));
    CUDA_CHECK(hipMalloc((void**)&dC, sizeof(double)*N));

    // Note the maximum size of threads in a block
    dim3 grid, threads;

    CUDA_CHECK(hipMemcpy(dA, hA, sizeof(double)*N, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dB, hB, sizeof(double)*N, hipMemcpyHostToDevice));

    //// Add the kernel call here
    vector_add<<<1,ThreadsInBlock>>>(x,dC, dA, dB, N);
    
    //printf("Pointer to device memory: %d",*dA);

    // Here we add an explicit synchronization so that we catch errors
    // as early as possible. Don't do this in production code!
    hipDeviceSynchronize();
    CHECK_ERROR_MSG("vector_add kernel");

    //// Copy back the results and free the device memory
    CUDA_CHECK(hipMemcpy(hC, dC, sizeof(double)*N, hipMemcpyDeviceToHost));


    for (int i = 0; i < N; i++)
        printf("%5.1f\n", hC[i]);

    CUDA_CHECK(hipFree(dA));
    CUDA_CHECK(hipFree(dB));
    CUDA_CHECK(hipFree(dC));
    free(x);
    return 0;
}
